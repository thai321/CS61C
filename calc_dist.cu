#include "hip/hip_runtime.h"
/*
 * Proj 3-2 SKELETON
 */

#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"



__global__ void transKernel(float *array1, float *array2, int width) {

    int current_index = (blockIdx.y*blockDim.y + threadIdx.y)* width +(blockIdx.x*blockDim.x + threadIdx.x);
    int replace = (blockIdx.x*blockDim.x + threadIdx.x)*width + blockIdx.y*blockDim.y + threadIdx.y;
    if (current_index< width*width) {
       array2[replace] = array1[current_index]; 
    }
}

void transpose(float *array1, float *array2, int width) {
    dim3 dim_blocks_per_grid(width/4, width/4);
    dim3 dim_threads_per_block(4, 4, 1);
    transKernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(array1, array2, width);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");
}

__global__ void flipKernel(float *array1, int width) {
    int current_index = blockIdx.x*blockDim.x + threadIdx.x;
    int replace = (width - 1 - current_index/width) * width + current_index % width;
    if (current_index < width * width / 2) {
        float temp = array1[current_index];
        array1[current_index] = array1[replace];
        array1[replace] = temp;
    }
}

void flip_vertical(float *array1, int width) {
    int threads_per_block = 512;
    int blocks_per_grid = width * width/2 / threads_per_block + 1;

    dim3 dim_blocks_per_grid(blocks_per_grid, 1);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);
    flipKernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(array1, width);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");
}

__global__ void distanceKernel(float *image, int i_width, float *temp, int t_width, float *blank) {
    int calc_index = (blockIdx.y*blockDim.y + threadIdx.y)*t_width + blockIdx.x*blockDim.x + threadIdx.x;
    float i = image[(blockIdx.y*blockDim.y + threadIdx.y)*i_width + blockIdx.x*blockDim.x + threadIdx.x];
    float j = temp[calc_index];
    blank[calc_index] = (i - j) * (i - j);
}

__global__ void reduceKernel(float *blank, int level, int width) {
    int current_index = (blockIdx.x * blockDim.x + threadIdx.x);
    current_index *=  2 * level;
    int valid = current_index + level;
    if (valid < width*width) {
        blank[current_index] += blank[valid];
    }
    
}

void findDistance(float *image, int i_width, float *temp, int t_width, float *blank) {

    dim3 dim_blocks_per_grid(t_width/16, t_width/16);
    dim3 dim_threads_per_block(16, 16, 1);
    distanceKernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(image, i_width, temp, t_width, blank);

    int threads = 512;
    int blocks = (t_width * t_width /threads) + 1;
    int level = 1;

    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");
    while (level != t_width * t_width) { 
       
        dim3 dim_blocks_per_grid(blocks, 1);
        dim3 dim_threads_per_block(threads, 1, 1);

        reduceKernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(blank, level, t_width);
        level *= 2;
        hipDeviceSynchronize();
        CUT_CHECK_ERROR("");

        if(blocks > 1) {
            blocks /= 2;
        } else {
            threads /= 2;
        }
    }
}

float calc_min_dist(float *image, int i_width, int i_height, float *temp, int t_width) {

    float min_dist = FLT_MAX;
    float current;
    float current_min = FLT_MAX;

    float *image2 = image;

    float *template_copy;
    CUDA_SAFE_CALL(hipMalloc(&template_copy, t_width * t_width * sizeof(float)));

    float *blank;
    CUDA_SAFE_CALL(hipMalloc(&blank, t_width * t_width * sizeof(float)));

    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, temp, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
    }
    image2 = image;

    if(current_min < min_dist)
    	min_dist = current_min;

    flip_vertical(temp, t_width); 
   
    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, temp, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
    }
    image2 = image;
    if(current_min < min_dist)
        min_dist = current_min;

    transpose(temp, template_copy, t_width); 

    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, template_copy, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
        
    }
    image2 = image;

    if(current_min < min_dist)
        min_dist = current_min;

    flip_vertical(template_copy, t_width);

    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, template_copy, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
        
    }
    image2 = image;
    if(current_min < min_dist)
        min_dist = current_min;

    transpose(template_copy, temp, t_width);

    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, temp, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
    }
    image2 = image;

    if(current_min < min_dist)
        min_dist = current_min;

    flip_vertical(temp, t_width);

    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, temp, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
        
    }
    image2 = image;
    if(current_min < min_dist)
        min_dist = current_min;

    transpose(temp, template_copy, t_width);

    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, template_copy, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
    }
    image2 = image;
    if(current_min < min_dist)
        min_dist = current_min;

    flip_vertical(template_copy, t_width); 
    current_min = FLT_MAX;
    for(int y = 0; y <= i_height - t_width; y++) {
        for(int x = 0; x <= i_width - t_width; x++) {
            findDistance(image2 + x, i_width, template_copy, t_width, blank);
            CUDA_SAFE_CALL(hipMemcpy(&current, blank, sizeof(float), hipMemcpyDeviceToHost));
            if(current < current_min)
                current_min = current;
        }
        image2 += i_width;
        
    }
    if(current_min < min_dist)
        min_dist = current_min;

    transpose(template_copy, temp, t_width);
    CUDA_SAFE_CALL(hipFree(template_copy));
    CUDA_SAFE_CALL(hipFree(blank));
    return min_dist;
}
